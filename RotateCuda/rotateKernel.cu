#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void rotateScatter(const float A[], float dst_array[], const float angle,
							  const int width, const int height)
{
}

int main()
{
	const char *pathname = "../Images/rectangle.raw";
	int width = 300;
	int height = 200;

	const int n = width * height;
	float A[n];
	float result[n];

	// Open input image and populate input array A.
	FILE *raw_p = fopen(pathname, "rb");
	if (raw_p)
	{
		fread(A, sizeof(float), n, raw_p);
	}

	// Modify input array A by normalizing values from 0->1.
	for (int i = 0; i < n; ++i)
	{
		A[i] = A[i] / 255.0;
	}

	// Call Kernel rotateScatter

	// Open output file and write result array.
	FILE *fp = fopen("test_image.raw", "wb");
	if (fp)
	{
		size_t r = fwrite(result, sizeof(result[0]), n, fp);
		printf("wrote %zu elements out of %d requested\n", r, n);
	}

	fclose(fp);
	fclose(raw_p);

	return 0;
}