#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void rotateScatter(float *A, float *dst_array, const float angle,
							  const int width, const int height)
{
	float c_x = width / 2.0;
	float c_y = height / 2.0;

	// Iterating horizontally through the image.
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{

			// Subtract center coordinates, so that we rotate with respect to the
			// center of the image.
			float x = j - c_x;
			float y = i - c_y;

			// Rotation operation
			float dst_x = cos(angle) * x - sin(angle) * y;
			float dst_y = sin(angle) * x + cos(angle) * y;

			// Add back the center "vector"
			dst_x = (int)(dst_x + c_x);
			dst_y = (int)(dst_y + c_y);

			// Check if the resulting point is inside the boundary of the image, i.e
			// 0->max_x, 0->max_y.
			if (dst_x >= 0 && dst_x < width && dst_y >= 0 && dst_y < height)
			{
				// If so then assign value from original array to dst_array at idx
				// location.
				int idx = dst_y * width + dst_x;
				dst_array[idx] = A[i * width + j];
			}
		}
	}
}

int main()
{
	const char *pathname = "../Images/rectangle.raw";
	int width = 300;
	int height = 200;

	const int n = width * height;
	float *A = (float *)malloc(sizeof(float) * n);
	float *R = (float *)malloc(sizeof(float) * n);
	float *d_a, *d_out;

	// Open input image and populate input array A.
	FILE *raw_p = fopen(pathname, "rb");
	if (raw_p)
	{
		fread(A, sizeof(float), n, raw_p);
	}

	// Modify input array A by normalizing values from 0->1.
	for (int i = 0; i < n; ++i)
	{
		A[i] = A[i] / 255.0;
	}

	// Allocate memory on device.
	hipMalloc((void **)&d_a, sizeof(float) * n);
	hipMalloc((void **)&d_out, sizeof(float) * n);

	// Copy Image array to device.
	hipMemcpy(d_a, A, sizeof(float) * n, hipMemcpyHostToDevice);

	// Call Kernel rotateScatter
	rotateScatter<<<1, 1>>>(d_a, d_out, M_PI / 4, width, height);

	hipMemcpy(R, d_out, sizeof(float) * n, hipMemcpyDeviceToHost);

	// Open output file and write result array.
	FILE *fp = fopen("test_image.raw", "wb");
	if (fp)
	{
		size_t r = fwrite(R, sizeof(R[0]), n, fp);
		printf("wrote %zu elements out of %d requested\n", r, n);
	}

	hipFree(d_a);
	hipFree(d_out);
	fclose(fp);
	fclose(raw_p);
	free(A);
	free(R);

	return 0;
}