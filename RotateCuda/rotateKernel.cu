#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void rotateScatter(float *A, float *dst_array, const float angle,
							  const int width, const int height)
{
	float c_x = width / 2.0;
	float c_y = height / 2.0;


	// Check if thread is in the range of the points. With width and height.
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Loop that check if each thread 

	float x = int(tid % width);
	float y = int(tid / height);

	// Subtract center coordinates, so that we rotate with respect to the
	// center of the image.
	x = x - c_x;
	y = y - c_y;

	// Rotation operation
	float dst_x = cos(angle) * x - sin(angle) * y;
	float dst_y = sin(angle) * x + cos(angle) * y;

	// Add back the center "vector"
	dst_x = (int)(dst_x + c_x);
	dst_y = (int)(dst_y + c_y);

	// Check if the resulting point is inside the boundary of the image, i.e
	// 0->max_x, 0->max_y.
	if (dst_x >= 0 && dst_x < width && dst_y >= 0 && dst_y < height)
	{
		// If so then assign value from original array to dst_array at idx
		// location.
		int idx = dst_y * width + dst_x;
		dst_array[idx] = A[tid];
	}
}

__global__ void rotateGather(float *A, float *dst_array, const float angle,
							 const int width, const int height)
{
	float c_x = width / 2.0;
	float c_y = height / 2.0;

	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	float x = int(tid % width);
	float y = int(tid / height);

	// Subtract center coordinates, so that we rotate with respect to the
	// center of the image.
	x = x - c_x;
	y = y - c_y;

	// Rotation operation
	float dst_x = cos(angle) * x - sin(angle) * y;
	float dst_y = sin(angle) * x + cos(angle) * y;

	// Add back the center "vector"
	dst_x = (int)(dst_x + c_x);
	dst_y = (int)(dst_y + c_y);

	// Check if the resulting point is inside the boundary of the image, i.e
	// 0->max_x, 0->max_y.
	if (dst_x >= 0 && dst_x < width && dst_y >= 0 && dst_y < height)
	{
		// If so then assign value from original array to dst_array at idx
		// location.
		int idx = dst_y * width + dst_x;
		dst_array[tid] = A[idx];
	}
}

int main(int argc, char *argv[])
{

	const char *pathname = "../Images/data_roberts.raw";
	int width = 300;
	int height = 200;

	if (argc == 3)
	{
		width = atoi(argv[1]);
		height = atoi(argv[2]);
	}

	const int n = width * height;
	float *A = (float *)malloc(sizeof(float) * n);
	float *R = (float *)malloc(sizeof(float) * n);

	memset(R, 0, n * sizeof(float));

	float *d_a, *d_out;

	// Open input image and populate input array A.
	FILE *raw_p = fopen(pathname, "rb");
	if (raw_p)
	{
		fread(A, sizeof(float), n, raw_p);
	}

	// Can create a kernel for this as well, or just add it in the rotation kernel.
	// Modify input array A by normalizing values from 0->1.
	for (int i = 0; i < n; ++i)
	{
		A[i] = A[i] / 255.0;
	}

	// Allocate memory on device.
	hipMalloc(&d_a, sizeof(float) * n);
	hipMalloc(&d_out, sizeof(float) * n);

	// Copy Image array to device.
	hipMemcpy(d_a, A, sizeof(float) * n, hipMemcpyHostToDevice);
	hipMemcpy(d_out, R, sizeof(float) * n, hipMemcpyHostToDevice);

	int NUM_THREADS = 1024;
	int NUM_BLOCKS = (int)ceil(n / NUM_THREADS);

	// // Call Kernel rotateScatter
	rotateScatter<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_out, M_PI / 4, width, height);
	// rotateGather<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_out, M_PI / 4, width, height);

	hipMemcpy(R, d_out, sizeof(float) * n, hipMemcpyDeviceToHost);

	// Open output file and write result array.
	FILE *fp = fopen("test_image.raw", "wb");
	if (fp)
	{
		size_t r = fwrite(R, sizeof(R[0]), n, fp);
		printf("wrote %zu elements out of %d requested\n", r, n);
	}

	hipFree(d_a);
	hipFree(d_out);
	// fclose(fp);
	// fclose(raw_p);
	free(A);
	free(R);

	return 0;
}