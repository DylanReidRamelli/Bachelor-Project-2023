#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Find max value in array of float.
float max(float input[], int size)
{
    float max_value = input[0];
    for (int i = 0; i < size; i++)
    {
        if (input[i] > max_value)
            max_value = input[i];
    }
    return max_value;
}

// Find min value in array of float.
float min(float input[], int size)
{
    float min_value = input[0];
    for (int i = 0; i < size; i++)
    {
        if (input[i] < min_value)
            min_value = input[i];
    }
    return min_value;
}

// 2D rotation.
void rotation(float coordinate[2], int idx, float angle)
{
    float tmp_x = coordinate[idx];
    float tmp_y = coordinate[idx + 1];
    coordinate[idx] = cos(angle) * tmp_x - sin(angle) * tmp_y;
    coordinate[idx + 1] = sin(angle) * tmp_x + cos(angle) * tmp_y;
}

// 2D corner rotation.
void rotateCorners(int output[2], int width, int height, float angle)
{

    float c_x = width / 2.0;
    float c_y = height / 2.0;
    float corners[8] = {
        0, 0, 0, (float)height, (float)width, 0, (float)width, (float)height};

    for (int j = 0; j < 8; j++)
    {
        if (j % 2 == 0)
        {
            corners[j] = corners[j] - c_x;
        }
        else
        {
            corners[j] = corners[j] - c_y;
        }
    }

    for (int i = 0; i < 8; i = i + 2)
    {
        // printf("nx:%f, ny:%f\n", corners[i], corners[i + 1]);
        rotation(corners, i, angle);
        // printf("nx:%f, ny:%f\n", corners[i], corners[i + 1]);
    }

    for (int j = 0; j < 8; j++)
    {
        if (j % 2 == 0)
        {
            corners[j] = corners[j] + c_x;
        }
        else
        {
            corners[j] = corners[j] + c_y;
        }
    }

    for (int j = 0; j < 8; j = j + 2)
    {
        // printf("nx:%f, ny:%f\n", corners[j], corners[j + 1]);
    }

    float x_values[] = {corners[0], corners[2], corners[4], corners[6]};
    float y_values[] = {corners[1], corners[3], corners[5], corners[7]};

    // for (int j = 0; j < 4; j++) {
    //   // printf("nx:%f\n", x_values[j]);
    // }

    int sizeX = ceil(max(x_values, 4) - min(x_values, 4));
    int sizeY = ceil(max(y_values, 4) - min(y_values, 4));

    // printf("sizex:%i, sizey:%i\n", sizeX, sizeY);

    output[0] = sizeX;
    output[1] = sizeY;
}

__global__ void rotateGatherNoLoss(float *A, float *dst_array, const float angle,
                                   int width, int height, int newSize[2])
{

    float c_x = width / 2.0;
    float c_y = height / 2.0;
    float c_x_out = newSize[0] / 2.0;
    float c_y_out = newSize[1] / 2.0;

    // Iterating horizontally through the image.
    for (int i = 0; i < newSize[1]; i++)
    {
        for (int j = 0; j < newSize[0]; j++)
        {

            // Subtract center coordinates, so that we rotate with respect to the
            // center of the image.
            float x = j - c_x_out;
            float y = i - c_y_out;

            // Rotation operation
            float dst_x = cos(angle) * x - sin(angle) * y;
            float dst_y = sin(angle) * x + cos(angle) * y;

            // Add back the center "vector"
            dst_x = (int)(dst_x + c_x);
            dst_y = (int)(dst_y + c_y);

            // Check if the resulting point is inside the boundary of the
            // image,i.e 0->max_x, 0->max_y.
            if (dst_x >= 0 && dst_x < width && dst_y >= 0 && dst_y < height)
            {
                // If so then assign value from original array to dst_array at idx
                // location.
                int idx = dst_y * width + dst_x;
                dst_array[i * newSize[0] + j] = A[idx];
            }
        }
    }
}

int main(int argc, char *argv[])
{

    const char *pathname = "../Images/data_roberts.raw";
    int width = 1303;
    int height = 2000;
    int iAngle = -145;
    char *oDataInfo;
    char *oDataPath;

    // if (argc == 3) {
    //   width = atoi(argv[1]);
    //   height = atoi(argv[2]);
    // }

    // if (argc == 4) {
    //   width = atoi(argv[1]);
    //   height = atoi(argv[2]);
    //   iDataPath = argv[3];
    // }

    if (argc == 4)
    {
        iAngle = atoi(argv[1]);
        oDataInfo = argv[2];
        oDataPath = argv[3];

        printf("ANGLE: %d, info: %s, output_path: %s\n", iAngle, oDataInfo,
               oDataPath);
    }

    const int n = width * height;
    float *A = (float *)malloc(sizeof(float) * n);

    float *d_a, *d_out;

    // Open input image and populate input array A.
    FILE *raw_p = fopen(pathname, "rb");
    if (raw_p)
    {
        fread(A, sizeof(float), n, raw_p);
    }

    // Can create a kernel for this as well, or just add it in the rotation kernel.
    // Modify input array A by normalizing values from 0->1.
    for (int i = 0; i < n; ++i)
    {
        A[i] = A[i] / 255.0;
    }

    const float ANGLE = M_PI / 3;
    int newSize[2] = {0, 0};

    // Rotate corners and get new dimentions of image.
    rotateCorners(newSize, width, height, ANGLE);

    int rSize = newSize[0] * newSize[1];

    float *R = (float *)malloc(sizeof(float) * rSize);
    memset(R, 0, rSize * sizeof(float));

    // Allocate memory on device.
    hipMalloc(&d_a, sizeof(float) * n);
    hipMalloc(&d_out, sizeof(float) * rSize);

    // Copy Image array to device.
    hipMemcpy(d_a, A, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_out, R, sizeof(float) * rSize, hipMemcpyHostToDevice);

    int NUM_THREADS = 256;
    int NUM_BLOCKS = (int)ceil(n / NUM_THREADS);

    hipMemcpy(R, d_out, sizeof(float) * rSize, hipMemcpyDeviceToHost);

    // Open output file and write result array.
    FILE *fpdata = fopen(oDataInfo, "w");
    if (fpdata)
    {
        fprintf(fpdata, "%i,%i", newSize[0], newSize[1]);
    }

    // Open output file and write result array.
    FILE *fp = fopen(oDataPath, "wb");
    if (fp)
    {
        size_t r = fwrite(R, sizeof(R[0]), newSize[0] * newSize[1], fp);
        printf("wrote %zu elements out of %d requested\n", r,
               newSize[0] * newSize[1]);
    }

    hipFree(d_a);
    hipFree(d_out);
    // fclose(fp);
    // fclose(raw_p);
    free(A);
    free(R);

    return 0;
}