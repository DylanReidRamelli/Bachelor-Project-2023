#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ float max(float *input, int size)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
}

__global__ void rotateGatherNoLoss(float *A, float *dst_array, const float angle,
                                   int width, int height, int mSize[2])
{

    int newSize[2] = {0, 0};

    // Rotate corners and get new dimentions of image.
    // rotateCorners(newSize, width, height, angle);

    float c_x = width / 2.0;
    float c_y = height / 2.0;
    float c_x_out = newSize[0] / 2.0;
    float c_y_out = newSize[1] / 2.0;

    printf("Size of new image: nx:%i, ny:%i\n", newSize[0], newSize[1]);

    // For python.
    mSize[0] = newSize[0];
    mSize[1] = newSize[1];

    dst_array = (float *)realloc(dst_array, mSize[0] * mSize[1] * sizeof(float));

    if (dst_array)
    {

        memset(dst_array, 0, mSize[0] * mSize[1] * sizeof(float));

        // Iterating horizontally through the image.
        for (int i = 0; i < newSize[1]; i++)
        {
            for (int j = 0; j < newSize[0]; j++)
            {

                // Subtract center coordinates, so that we rotate with respect to the
                // center of the image.
                float x = j - c_x_out;
                float y = i - c_y_out;

                // Rotation operation
                float dst_x = cos(angle) * x + sin(angle) * y;
                float dst_y = -sin(angle) * x + cos(angle) * y;

                // Add back the center "vector"
                dst_x = (int)(dst_x + c_x);
                dst_y = (int)(dst_y + c_y);

                // Check if the resulting point is inside the boundary of the
                // image,i.e 0->max_x, 0->max_y.
                if (dst_x >= 0 && dst_x < width && dst_y >= 0 && dst_y < height)
                {
                    // If so then assign value from original array to dst_array at idx
                    // location.
                    int idx = dst_y * width + dst_x;
                    dst_array[i * mSize[0] + j] = A[idx];
                }
            }
        }
    }
}

int main()
{

    const char *pathname = "../Images/data_rectangle.raw";
    int width = 300;
    int height = 200;

    const int n = width * height;
    float *A = (float *)malloc(sizeof(float) * n);
    float *R = (float *)malloc(sizeof(float) * n);

    memset(R, 0, n * sizeof(float));

    float *d_a, *d_out;

    // Open input image and populate input array A.
    FILE *raw_p = fopen(pathname, "rb");
    if (raw_p)
    {
        fread(A, sizeof(float), n, raw_p);
    }

    // Can create a kernel for this as well, or just add it in the rotation kernel.
    // Modify input array A by normalizing values from 0->1.
    for (int i = 0; i < n; ++i)
    {
        A[i] = A[i] / 255.0;
    }

    // Allocate memory on device.
    hipMalloc(&d_a, sizeof(float) * n);
    hipMalloc(&d_out, sizeof(float) * n);

    // Copy Image array to device.
    hipMemcpy(d_a, A, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_out, R, sizeof(float) * n, hipMemcpyHostToDevice);

    int NUM_THREADS = 256;
    int NUM_BLOCKS = (int)ceil(n / NUM_THREADS);

    hipMemcpy(R, d_out, sizeof(float) * n, hipMemcpyDeviceToHost);

    // Open output file and write result array.
    FILE *fp = fopen("test_image.raw", "wb");
    if (fp)
    {
        size_t r = fwrite(R, sizeof(R[0]), n, fp);
        printf("wrote %zu elements out of %d requested\n", r, n);
    }

    hipFree(d_a);
    hipFree(d_out);
    // fclose(fp);
    // fclose(raw_p);
    free(A);
    free(R);

    return 0;
}