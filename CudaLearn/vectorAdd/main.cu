#include <iostream>
#include <hip/hip_runtime.h>

__global__ void RotateArray(int*originalA, int*newA, int degrees){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * threadIdx.y;

    
}

__global__ void AddIntsCUDA(int *a, int *b){
    a[0] += b[0];
}


int main(int argc, char const *argv[])
{

    int a = 5,b = 9;
    int *d_a,*d_b;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);


    AddIntsCUDA<<<1,1>>>(d_a,d_b);


    hipMemcpy(&a, d_a,sizeof(int), hipMemcpyDeviceToHost);



    std::cout << "The answer is:" << a <<std::endl;

    hipFree(d_a);
    hipFree(d_b);

    
    return 0;
}
